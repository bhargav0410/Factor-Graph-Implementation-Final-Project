#include "hip/hip_runtime.h"
#include <iostream>
#include "factor_graph.h"
#include "ldpc_bp.h"
#include <cstdlib>
#include <chrono>
#include <ctime>
#include <cmath>
#include <algorithm>
#include <random>
#include "ldpc_bp_cuda.cuh"

using namespace std::chrono;

int main (int argc, char* argv[]) {
    //To measure execution time
    double serial_time = 1e30;
    duration<double> timediff;
    high_resolution_clock::time_point start, finish;
    float snr = 10; //SNR in dB

    int n = 12, m = 1, k = 4, iter = 10, num_syms = std::max(1024,n);
    float rate = 0;
    if (argc >= 4) {
        n = atoi(argv[1]);
        m = atoi(argv[2]);
        k = atoi(argv[3]);
        if (argc >= 5) {
            snr = strtof(argv[4], NULL);
        }
        if (argc >= 6) {
            iter = atoi(argv[5]);
        }
        if (argc >= 7) {
            num_syms = atoi(argv[6]);
        }
    }

    std::cout << "Creating parity check matrix...\n";
    ldpc_bp_cuda ldpc;
    start = high_resolution_clock::now();
    ldpc.create_H_mat(n, m, k);
    finish = high_resolution_clock::now();
    timediff = duration_cast<duration<double>>(finish - start);
    serial_time = std::min(serial_time, timediff.count());
    std::cout << "Time: " << serial_time << std::endl;

    std::cout << "Anticipated Rate >= " << ldpc.getRate() << "\n";
    std::cout << "Creating generator matrix...\n";
    serial_time = 1e30;
    start = high_resolution_clock::now();
    ldpc.gen_mat_from_H_mat();
    finish = high_resolution_clock::now();
    timediff = duration_cast<duration<double>>(finish - start);
    serial_time = std::min(serial_time, timediff.count());
    std::cout << "Time: " << serial_time << std::endl;

    std::cout << "Converting to standard form...\n";
    serial_time = 1e30;
    start = high_resolution_clock::now();
    ldpc.standard_form();
    finish = high_resolution_clock::now();
    timediff = duration_cast<duration<double>>(finish - start);
    serial_time = std::min(serial_time, timediff.count());
    std::cout << "Time: " << serial_time << std::endl;

    ldpc.H_mat_comp_form();
    ldpc.create_list_from_mat();
    ldpc.check_matrices();

    //Creating input random vector and encoding it
    std::vector<int> in, out;
    srand(time(NULL));
    for (int i = 0; i < num_syms*ldpc.get_num_input_syms(); i++) {
        in.push_back(rand()%2);
    }
    
    print_vector(in);
    ldpc.print_matrices();
    ldpc.encode_using_G_mat_cuda(in, out);
    std::cout << "Encoding done...\n";
    if (ldpc.check_vector(out) != 0) {
        std::cout << "Encoding incorrect...\n";
    }
    print_vector(out);
    std::cout << "Final Rate = " << ldpc.getGenMatRate() << "\n";

    //Noise generation (equivalent to passing through a channel)

    //Noise variance based on input SNR
    float std_dev = pow((float)10.0, -((float)snr/(float)10.0));
    std::cout << "Noise power: " << std_dev << "\n";
    std::default_random_engine generator;
    std::normal_distribution<float> distribution(0.0, std_dev);
    std::vector<float> awgn(out.size()), chan_in(out.size()), chan_out(out.size());

    //Passing through AWGN channel
    for (int i = 0; i < awgn.size(); i++) {
        //Creating vector ready to be transmitted through channel
        chan_in[i] = 2*(float)out[i] - 1;
        //Creating noise for channel emulation
        awgn[i] = distribution(generator);
        //Passing through AWGN channel
        chan_out[i] = chan_in[i] + awgn[i];
    }
    print_vector(chan_in);
    print_vector(chan_out);

    std::vector<int> final_out;

    //Decode noise signal
    ldpc.sum_product_decoding_cuda(chan_out, final_out, snr, iter);
    print_vector(in);
    print_vector(final_out);

    float ber = 0;
    for (int i = 0; i < final_out.size(); i++) {
        ber += abs(in[i] - final_out[i]);
    }
    std::cout << "BER: " << ber/(float)final_out.size() << "\n";

    //ldpc.gen_mat_from_H_mat();

    //std::cin.get();
    return 0;

}