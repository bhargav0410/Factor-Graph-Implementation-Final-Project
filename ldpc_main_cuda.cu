#include "hip/hip_runtime.h"
#include <iostream>
#include "factor_graph.h"
#include "ldpc_bp.h"
#include <cstdlib>
#include <chrono>
#include <ctime>
#include <cmath>
#include <algorithm>
#include <random>
#include <iostream>
#include <string>
#include "ldpc_bp_cuda.cuh"

using namespace std::chrono;

int main (int argc, char* argv[]) {
    //To measure execution time
    double serial_encode = 0, cuda_encode = 0, serial_decode = 0, cuda_decode = 0;
    duration<double> timediff;
    high_resolution_clock::time_point start, finish;
    float snr = 10; //SNR in dB

    int n = 12, m = 1, k = 4, iter = 10, num_syms = 1, num_times = 1;
    float rate = 0;
    if (argc >= 4) {
        n = atoi(argv[1]);
        m = atoi(argv[2]);
        k = atoi(argv[3]);
        if (argc >= 5) {
            snr = strtof(argv[4], NULL);
        }
        if (argc >= 6) {
            iter = atoi(argv[5]);
        }
        if (argc >= 7) {
            num_syms = atoi(argv[6]);
        }
        if (argc >= 8) {
            num_times = atoi(argv[7]);
        }
    }

    double serial_construct = 0, cuda_construct = 0;

    for (int times = 0; times < num_times; times++) {
        std::cout << "Creating parity check matrix...\n";
        ldpc_bp_cuda ldpc;
        ldpc.create_H_mat(n, m, k);

        std::cout << "Anticipated Rate >= " << ldpc.getRate() << "\n";
        std::cout << "Creating generator matrix...\n";
        start = high_resolution_clock::now();
        ldpc.gen_mat_from_H_mat();
        finish = high_resolution_clock::now();
        serial_construct += duration_cast<duration<double>>(finish - start).count();
        cuda_construct += ldpc.gen_mat_from_H_mat_cu();
        std::cout << "Converting to standard form...\n";
        ldpc.standard_form();
        ldpc.H_mat_comp_form();
        ldpc.create_list_from_mat();
        ldpc.check_matrices();

        //Creating input random vector and encoding it
        std::vector<int> in(num_syms*ldpc.get_num_input_syms()), out;
        srand(time(NULL));
        for (int i = 0; i < num_syms*ldpc.get_num_input_syms(); i++) {
            in[i] = (rand()%2);
        }
        
        //print_vector(in);
        //ldpc.print_matrices();
        start = high_resolution_clock::now();
        ldpc.encode_using_G_mat(in, out);
        finish = high_resolution_clock::now();
        serial_encode += duration_cast<duration<double>>(finish - start).count();
        cuda_encode += ldpc.encode_using_G_mat_cuda(in, out);
        std::cout << "Encoding done...\n";
        if (ldpc.check_vector(out) != 0) {
            std::cout << "Encoding incorrect...\n";
        }
        //print_vector(out);
        std::cout << "Final Rate = " << ldpc.getGenMatRate() << "\n";

        //Noise generation (equivalent to passing through a channel)

        //Noise variance based on input SNR
        float std_dev = pow((float)10.0, -((float)snr/(float)10.0));
        std::cout << "Noise power: " << std_dev << "\n";
        std::default_random_engine generator;
        std::normal_distribution<float> distribution(0.0, std_dev);
        std::vector<float> awgn(out.size()), chan_in(out.size()), chan_out(out.size());

        //Passing through AWGN channel
        for (int i = 0; i < awgn.size(); i++) {
            //Creating vector ready to be transmitted through channel
            chan_in[i] = 2*(float)out[i] - 1;
            //Creating noise for channel emulation
            awgn[i] = distribution(generator);
            //Passing through AWGN channel
            chan_out[i] = chan_in[i] + awgn[i];
        }
        //print_vector(chan_in);
        //print_vector(chan_out);

        std::vector<int> final_out;

        //Decode noisy signal
        start = high_resolution_clock::now();
        ldpc.sum_product_decode(chan_out, final_out, snr, iter);
        finish = high_resolution_clock::now();
        final_out.clear();
        serial_decode += duration_cast<duration<double>>(finish - start).count();
        cuda_decode += ldpc.sum_product_decoding_cuda(chan_out, final_out, snr, iter);
        //print_vector(in);
        //print_vector(final_out);

        float ber = 0;
        for (int i = 0; i < final_out.size(); i++) {
            ber += abs(in[i] - final_out[i]);
        }
        std::cout << "BER: " << ber/(float)final_out.size() << "\n";
    }
    printf("Serial construction time: %f secs\n", serial_construct);
    printf("Serial encoding time: %f secs\n", serial_encode);
    printf("Serial decoding time: %f secs\n", serial_decode);
    printf("CUDA construction time: %f secs\n", cuda_construct);
    printf("CUDA encoding time: %f secs\n", cuda_encode);
    printf("CUDA decoding time: %f secs\n", cuda_decode);

    std::ofstream ofs;
    std::string file = "cuda_time_" + std::to_string(n) + "_" + std::to_string(m) + "_" + std::to_string(k) + "_" + std::to_string(num_syms) + "_syms_" + std::to_string(num_times) + "_times.csv";
    ofs.open (file.c_str(), std::ofstream::out | std::ofstream::trunc);
    ofs << serial_construct << ",";
    ofs << serial_encode << ",";
    ofs << serial_decode << "\n";
    ofs << cuda_construct << ",";
    ofs << cuda_encode << ",";
    ofs << cuda_decode << "\n";
    ofs.close();
    return 0;

}